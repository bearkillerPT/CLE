#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

__global__ static void gaussianEliminationRows(double *all_matrices, int matrix_order, int matrices_count);
__device__ static void find_matrix_non_zero_row(double *matrix, int row_index, int column_index, int matrix_order, int *row);
__device__ static void swap_matrix_rows(double *matrix, int first_row, int second_row, int thread_index, int matrix_order);

int main(int argc, char const *argv[])
{
    // usage
    if (argc == 1)
    {
        printf("Program Usage:\n\t ./ex2 (matrix*.txt)+\n");
        return 1;
    }

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // time vars
    double exec_start = seconds();
    for (int matrix_file_i = 1; matrix_file_i < argc; matrix_file_i++)
    {
        // for every file
        FILE *matrix_file = fopen(argv[matrix_file_i], "r");
        if (matrix_file != NULL)
        {
            int matrix_count, matrix_order;
            fread(&matrix_count, sizeof(int), 1, matrix_file);
            fread(&matrix_order, sizeof(int), 1, matrix_file);
            int all_matrices_size = matrix_count * matrix_order * matrix_order * sizeof(double);
            double *all_matrices = (double *)malloc(all_matrices_size);

            // read all matrices
            for (int matrix_i = 0; matrix_i < matrix_count; matrix_i++)
            {
                double *current_matrix = &all_matrices[matrix_i * matrix_order * matrix_order];
                for (int i = 0; i < matrix_order * matrix_order; i++)
                    fread(&current_matrix[i], sizeof(double), 1, matrix_file);
            }

            fclose(matrix_file);
            double *gpu_matrices;

            // allocate memory for the matrices on the gpu and memcopy
            CHECK(hipMalloc((void **)&gpu_matrices, all_matrices_size));
            CHECK(hipMemcpy(gpu_matrices, all_matrices, all_matrices_size, hipMemcpyHostToDevice));

            dim3 grid(matrix_count, 1, 1);
            dim3 block(matrix_order, 1, 1);
            gaussianEliminationRows<<<grid, block>>>(gpu_matrices, matrix_order, matrix_count);

            // error check
            CHECK(hipGetLastError());

            // transfering the matrices back
            CHECK(hipMemcpy(all_matrices, gpu_matrices, all_matrices_size, hipMemcpyDeviceToHost));

            // determinant calculation for each matrix
            for (int i = 0; i < matrix_count; i++)
            {
                double determinant = 1.0;
                int current_matrix_offset = i * matrix_order * matrix_order;
                for (int j = 0; j < matrix_order; j++)
                    determinant *= all_matrices[matrix_order * j + j + current_matrix_offset];
                fprintf(stdout, "\nMatrix %d: %E", (i + 1), determinant);
            }

            // memory frees and gpu device reset
            free(all_matrices);
            CHECK(hipFree(gpu_matrices));
            CHECK(hipDeviceReset());
        }
        else
            printf("The provided file: %s\n couldn't be opened!\n", argv[matrix_file_i]);
    }

    // execution time
    double exec_time = seconds() - exec_start;
    fprintf(stdout, "\n\nTime Elapsed: %f s\n", exec_time);
    return 0;
}

__global__ static void gaussianEliminationRows(double *all_matrices, int matrix_order, int matrices_count)
{
    int matrix_i = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
    int thread_ID = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
    if (matrix_i >= matrices_count)
        return;
    double *current_matrix = &all_matrices[matrix_i * matrix_order * matrix_order];

    int current_row_pivot = 0;
    int current_column_pivot = 0;
    while (current_row_pivot < matrix_order && current_column_pivot < matrix_order)
    {
        if (current_matrix[current_row_pivot * matrix_order + current_column_pivot] == 0)
        {
            int row = -1;
            find_matrix_non_zero_row(current_matrix, current_row_pivot, current_column_pivot, matrix_order, &row);

            __syncthreads();

            // if there is no non zero row just set the value to 0 so that the determinant will also be 0 and return
            if (row == -1)
                return;

            // swap rows
            if (row != current_row_pivot)
                swap_matrix_rows(current_matrix, current_row_pivot, row, thread_ID, matrix_order);

            __syncthreads();
        }

        // Calculate the gaussian elimination factor first
        for (int next_row = current_row_pivot + 1; next_row < matrix_order; next_row++)
        {
            double factor;
            if (thread_ID < matrix_order && thread_ID >= current_column_pivot)
            {
                // matrix[i][i]
                int pivot_index = current_row_pivot * matrix_order + current_column_pivot;
                // matrix[k][i]
                int next_row_index = next_row * matrix_order + current_column_pivot;
                // matrix[k][i] / matrix[i][i]
                factor = current_matrix[next_row_index] / current_matrix[pivot_index];
            }

            __syncthreads();

            // apply the transformation for the remaining elements in the same row on the next columns

            if (thread_ID < matrix_order && thread_ID >= current_column_pivot)
            {
                // matrix[k][j]
                int thread_row_index = next_row * matrix_order + thread_ID;
                // matrix[i][j]
                int factor_element_index = current_row_pivot * matrix_order + thread_ID;
                // matrix[k][j] = matrix[k][j] - (matrix[k][i] / matrix[i][i]) * matrix[i][j]
                current_matrix[thread_row_index] -= factor * current_matrix[factor_element_index];
            }
        }
        // wait and move to the next pivot
        __syncthreads();

        current_row_pivot++;
        current_column_pivot++;
    }
}

// Function for a thread to find a non zero row
__device__ static void find_matrix_non_zero_row(double *matrix, int row_index, int column_index, int matrix_order, int *row)
{
    *row = -1;
    for (int i = row_index; i < matrix_order; i++)
    {
        if (matrix[i * matrix_order + column_index] != 0.0)
        {
            *row = i;
            break;
        }
    }
}
__device__ static void swap_matrix_rows(double *matrix, int first_row, int second_row, int thread_index, int matrix_order)
{
    int first_row_index = first_row * matrix_order + thread_index;
    int second_row_index = second_row * matrix_order + thread_index;
    double temp = matrix[first_row_index];
    matrix[first_row_index] = matrix[second_row_index];
    matrix[second_row_index] = temp;
}